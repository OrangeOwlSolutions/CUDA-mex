#include "hip/hip_runtime.h"
#include <stdio.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "mex.h"

#include "Utilities.cuh"

#define BLOCKSIZE	512

/*******************/
/* SQUARING KERNEL */
/*******************/
__global__ void squareKernel(double * __restrict__ d_vec, const int N) {

	const int tid = threadIdx.x + blockIdx.x * blockDim.x;

	if (tid >= N) return;

	d_vec[tid] = d_vec[tid] * d_vec[tid];
}

/****************/
/* MEX FUNCTION */
/****************/
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {

	// --- Recovering the pointer to the input host variable
	double *h_input = mxGetPr(prhs[0]);

	// --- Recovering the number of elements of the input variable (the input variable can be also a matrix)
	int numElements = mxGetN(prhs[0]) * mxGetM(prhs[0]);

	// --- Allocating space for the input/output device variable
	double *d_vec; gpuErrchk(hipMalloc(&d_vec, numElements * sizeof(double)));
	
	// --- Moving the input from host to device
	gpuErrchk(hipMemcpy(d_vec, h_input, numElements * sizeof(double), hipMemcpyHostToDevice));

	squareKernel<<<iDivUp(numElements, BLOCKSIZE), BLOCKSIZE>>>(d_vec, numElements);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());

	// --- Allocating space for the output output host variable
	plhs[0] = mxCreateDoubleMatrix(1, numElements, mxREAL);
	
	// --- Recovering the pointer to the output host variable
	double *h_output = mxGetPr(plhs[0]);

	gpuErrchk(hipMemcpy(h_output, d_vec, numElements * sizeof(double), hipMemcpyDeviceToHost));

	gpuErrchk(hipFree(d_vec));

}
